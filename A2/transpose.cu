#include "hip/hip_runtime.h"
#include <iostream>
#include <chrono>
#include <sys/time.h>
#include <hip/hip_runtime.h>

#include "hip/hip_runtime_api.h"
#include <string>

#define EXPERIMENTS 4

#ifndef DTYPE
#define DTYPE float
#endif

#ifndef TILE_DIM
#define TILE_DIM 32
#endif

#ifndef BLOCK_ROWS
#define BLOCK_ROWS 8
#endif

#ifndef NUM_REPS
#define NUM_REPS 100
#endif

// #define PRETTY_PRINT

#define INIT_CUDA_TIMER      \
    hipEvent_t start, stop; \
    hipEventCreate(&start); \
    hipEventCreate(&stop);

#define START_CUDA_TIMER \
    hipEventRecord(start);

#define STOP_CUDA_TIMER                       \
    checkCudaErrors(hipDeviceSynchronize()); \
    hipEventRecord(stop);                    \
    hipEventSynchronize(stop);

#define GET_CUDA_ELAPSED_TIME                        \
    ({                                               \
        float runtime = 0.0f;                        \
        hipEventElapsedTime(&runtime, start, stop); \
        runtime;                                     \
    })

#define DELETE_CUDA_TIMER        \
    if (start)                   \
        hipEventDestroy(start); \
    if (stop)                    \
        hipEventDestroy(stop);

/**
 * Simple copy reference kernel
 *
 * Arguments:
 * - `idata`: source matrix defined in a single pointer.
 * - `odata`: destination matrix, can be pre-filled, defined in a single pointer.
 */
__global__ void copy(const DTYPE *idata, DTYPE *odata)
{
    int x = blockIdx.x * TILE_DIM + threadIdx.x;
    int y = blockIdx.y * TILE_DIM + threadIdx.y;
    int width = gridDim.x * TILE_DIM;

    for (int i = 0; i < TILE_DIM; i += BLOCK_ROWS)
    {
        odata[(y + i) * width + x] = idata[(y + i) * width + x];
    }
}

/**
 * Naive implementation of matrix transpose with memory coalescing.
 * Global memory reads are coalesced but writes are not.
 *
 * @param idata: input matrix (single pointer)
 * @param odata: output matrix (single pointer)
 * @param width: matrix width
 * @param height: matrix height
 *
 * @note Source: [NVidia Paper](https://www.cs.colostate.edu/~cs675/MatrixTranspose.pdf)
 * @note Source: [NVidia cuda-samples](https://github.com/NVIDIA-developer-blog/code-samples/blob/master/series/cuda-cpp/transpose/transpose.cu#L98)
 */
__global__ void transposeNaive(const DTYPE *idata, DTYPE *odata)
{
    int x = blockIdx.x * TILE_DIM + threadIdx.x;
    int y = blockIdx.y * TILE_DIM + threadIdx.y;
    int width = gridDim.x * TILE_DIM;

    for (int i = 0; i < TILE_DIM; i += BLOCK_ROWS)
    {
        odata[width * x + (y + i)] = idata[(y + i) * width + x];
    }
}

/**
 * Use shared memory to achieve coalesing in both
 * reads and writes.
 *
 * @param idata:    
 * @param odata:
 *
 * @note Source: [NVidia cuda-samples](https://github.com/NVIDIA-developer-blog/code-samples/blob/master/series/cuda-cpp/transpose/transpose.cu#L111)
 */
__global__ void transposeCoalesced(const float *idata, float *odata)
{
    __shared__ float tile[TILE_DIM][TILE_DIM];

    int x = blockIdx.x * TILE_DIM + threadIdx.x;
    int y = blockIdx.y * TILE_DIM + threadIdx.y;
    int width = gridDim.x * TILE_DIM;

    for (int i = 0; i < TILE_DIM; i += BLOCK_ROWS)
    {
        tile[threadIdx.y + i][threadIdx.x] = idata[(y + i) * width + x];
    }

    __syncthreads();
    // block offset
    x = blockIdx.y * TILE_DIM + threadIdx.x;
    y = blockIdx.x * TILE_DIM + threadIdx.y;

    for (int i = 0; i < TILE_DIM; i += BLOCK_ROWS)
    {
        odata[(y + i) * width + x] = tile[threadIdx.x][threadIdx.y + i];
    }
}

__global__ void transposeCoalescedNoBankConflicts(const float *idata, float *odata)
{
    __shared__ float tile[TILE_DIM][TILE_DIM + 1];

    int x = blockIdx.x * TILE_DIM + threadIdx.x;
    int y = blockIdx.y * TILE_DIM + threadIdx.y;
    int width = gridDim.x * TILE_DIM;

    for (int i = 0; i < TILE_DIM; i += BLOCK_ROWS)
    {
        tile[threadIdx.y + i][threadIdx.x] = idata[(y + i) * width + x];
    }

    __syncthreads();
    // block offset
    x = blockIdx.y * TILE_DIM + threadIdx.x;
    y = blockIdx.x * TILE_DIM + threadIdx.y;

    for (int i = 0; i < TILE_DIM; i += BLOCK_ROWS)
    {
        odata[(y + i) * width + x] = tile[threadIdx.x][threadIdx.y + i];
    }
}

double effectiveBandWidthSquaredMatrixTranspose(const unsigned int dim, double time)
{
    return 2 * (dim * dim) * sizeof(float) * 1e-6 * NUM_REPS / (time);
}

DTYPE mError(unsigned int DIM, const DTYPE *A, const DTYPE *B)
{
    int i, j;

    DTYPE error = static_cast<DTYPE>(0);
    for (i = 0; i < DIM; i++)
    {
        for (j = 0; j < DIM; j++)
        {
            error += fabs(A[i * DIM + j] - B[i * DIM + j]);
        }
    }

    return error;
}

int main(int argc, char *argv[])
{

    // INITIALIZE MATRIX SHAPE
    size_t DIM = 2;
    if (argc == 1)
    {
#ifdef PRETTY_PRINT
        std::cout
            << "No arguments" << std::endl
            << "\tNote that this works only with squared matrices." << std::endl;
#endif
    }

    if (argc > 1)
    {
        int exponent = std::stoi(argv[1]);
        if (exponent < 1)
        {
#ifdef PRETTY_PRINT
            std::cerr << "ERROR: exponent must be > 1!" << std::endl;
#endif
            return EXIT_FAILURE;
        }
        if (exponent > 14)
        {
#ifdef PRETTY_PRINT
            std::cerr << "ERROR: dude, you seriusly want to allocate more than 4GB of data?" << std::endl;
#endif
            return EXIT_FAILURE;
        }
        DIM = DIM << exponent;
    }
#ifdef PRETTY_PRINT
    printf("Matrix has size: [%zu x %zu]\n", DIM, DIM);
#endif
    // VERIFY DEFINED VARIABLES
    if (DIM % TILE_DIM)
    {
#ifdef PRETTY_PRINT
        std::cerr << "Matrix shape: [" << DIM << ", " << DIM << "] must be a multiple of TILE_DIM: " << TILE_DIM << std::endl;
#endif
        return EXIT_FAILURE;
    }

    // DEFINE MATRIX
    size_t mem_size = DIM * DIM * sizeof(DTYPE);

    DTYPE *data_cpu = (DTYPE *)malloc(mem_size);
    DTYPE *reference = (DTYPE *)malloc(mem_size);
    DTYPE *result_cpu = (DTYPE *)malloc(mem_size);
    DTYPE *data_gpu, *result_gpu;

    unsigned int repetitions = 0;
    double time = 0.0;

    const char *experimentsNames[EXPERIMENTS];
    DTYPE errors[EXPERIMENTS];
    double bandwidths[EXPERIMENTS];

    hipMalloc(&data_gpu, mem_size);
    hipMalloc(&result_gpu, mem_size);

    // INITIALIZE CPU MATRIX
    for (size_t i = 0; i < DIM; ++i)
    {
        for (size_t j = 0; j < DIM; ++j)
        {
            data_cpu[i * DIM + j] = static_cast<DTYPE>(rand()) / static_cast<DTYPE>(RAND_MAX);
        }
    }

    for (size_t i = 0; i < DIM; i++)
    {
        for (size_t j = 0; j < DIM; j++)
        {
            reference[j * DIM + i] = data_cpu[i * DIM + j];
        }
    }

    // COPY DATA TO GPU
    hipMemcpy(data_gpu, data_cpu, mem_size, hipMemcpyHostToDevice);
    hipMemset(result_gpu, 0, mem_size);
#ifdef PRETTY_PRINT
    printf("======================================= Device properties ========================================\n");
#endif
    int deviceCount = 0;
    hipError_t error_id = hipGetDeviceCount(&deviceCount);

    if (error_id != hipSuccess)
    {
#ifdef PRETTY_PRINT
        printf("Result = FAIL\n");
#endif
        exit(EXIT_FAILURE);
    }

    if (deviceCount == 0)
    {
#ifdef PRETTY_PRINT
        printf("No CUDA enabled devices available. Exiting");
#endif
        exit(EXIT_FAILURE);
    }
    else
    {
#ifdef PRETTY_PRINT
        printf("Detected %d CUDA Capable device(s)\n", deviceCount);
        printf("Using device 0\n");
#endif
    }

    hipSetDevice(0);
    hipDeviceProp_t deviceProp;
    hipGetDeviceProperties(&deviceProp, 0);

    // Calculate memory bandwidth
    int mem_clock_rate = deviceProp.memoryClockRate; // kHz
    int mem_width = deviceProp.memoryBusWidth;       // bits
    // 2 * deviceProp.memoryClockRate * (deviceProp.memoryBusWidth/8)/ 1.0e6;
    double mem_bandwidth = (double)mem_clock_rate * 1e3 * (mem_width / 8) * 2 / 1e9;

#ifdef PRETTY_PRINT
    printf("Memory clock rate: %d MHz\n", mem_clock_rate / 1000);
    printf("Memory width: %d bits\n", mem_width);
    printf("Theoretical memory bandwidth: %f GBps\n", mem_bandwidth);
#endif
    // printf("====================================== Problem computations ======================================\n");
    // SET GRID AND BLOCK DIMENSIONS
    dim3 dim_grid(DIM / TILE_DIM, DIM / TILE_DIM, 1);
    dim3 dim_block(TILE_DIM, BLOCK_ROWS, 1);

    // ############# COPY (reference)
    // warm up
    copy<<<dim_grid, dim_block>>>(data_gpu, result_gpu);
    INIT_CUDA_TIMER;
    hipMemset(result_gpu, 0, mem_size);
    START_CUDA_TIMER;
    for (repetitions = 0; repetitions < NUM_REPS; ++repetitions)
    {
        copy<<<dim_grid, dim_block>>>(data_gpu, result_gpu);
    }
    STOP_CUDA_TIMER;

    time = GET_CUDA_ELAPSED_TIME;
    experimentsNames[0] = "Copy - reference";
    bandwidths[0] = effectiveBandWidthSquaredMatrixTranspose(DIM, time);
    errors[0] = 0.0f;

    // ############# NAIVE 2
    transposeNaive<<<dim_grid, dim_block>>>(data_gpu, result_gpu);
    hipMemset(result_gpu, 0, mem_size);
    START_CUDA_TIMER;
    for (repetitions = 0; repetitions < NUM_REPS; ++repetitions)
    {
        transposeNaive<<<dim_grid, dim_block>>>(data_gpu, result_gpu);
    }
    STOP_CUDA_TIMER;

    checkCudaErrors(hipMemcpy(result_cpu, result_gpu, mem_size, hipMemcpyDeviceToHost));

    time = GET_CUDA_ELAPSED_TIME;
    experimentsNames[1] = "Transpose Naive";
    bandwidths[1] = effectiveBandWidthSquaredMatrixTranspose(DIM, time);
    errors[1] = mError(DIM, result_cpu, reference);

    // ############# SHARED 1 COALESCED
    transposeCoalesced<<<dim_grid, dim_block>>>(data_gpu, result_gpu);
    hipMemset(result_gpu, 0, mem_size);
    START_CUDA_TIMER;
    for (repetitions = 0; repetitions < NUM_REPS; ++repetitions)
    {
        transposeCoalesced<<<dim_grid, dim_block>>>(data_gpu, result_gpu);
    }
    STOP_CUDA_TIMER;

    checkCudaErrors(hipMemcpy(result_cpu, result_gpu, mem_size, hipMemcpyDeviceToHost));

    time = GET_CUDA_ELAPSED_TIME;
    experimentsNames[2] = "Transpose Shared Coalesced";
    bandwidths[2] = effectiveBandWidthSquaredMatrixTranspose(DIM, time);
    errors[2] = mError(DIM, result_cpu, reference);

    // ############# SHARED 2 COALESCED
    transposeCoalescedNoBankConflicts<<<dim_grid, dim_block>>>(data_gpu, result_gpu);
    hipMemset(result_gpu, 0, mem_size);
    START_CUDA_TIMER;
    for (repetitions = 0; repetitions < NUM_REPS; ++repetitions)
    {
        transposeCoalescedNoBankConflicts<<<dim_grid, dim_block>>>(data_gpu, result_gpu);
    }
    STOP_CUDA_TIMER;

    checkCudaErrors(hipMemcpy(result_cpu, result_gpu, mem_size, hipMemcpyDeviceToHost));

    time = GET_CUDA_ELAPSED_TIME;
    experimentsNames[3] = "Transpose Naive Coalesced No Bank Conflicts";
    bandwidths[3] = effectiveBandWidthSquaredMatrixTranspose(DIM, time);
    errors[3] = mError(DIM, result_cpu, reference);

// ############# Final Print
#ifdef PRETTY_PRINT
    printf("====================================== Results ======================================\n");
    printf("%40s\t%25s\t%25s\n", "experiments", "Bandwidth [GBps]", "Errors #");

    for (int i = 0; i < EXPERIMENTS; i++)
    {
        printf("%40s\t%20.5f\t%20.5f\n", experimentsNames[i], bandwidths[i], errors[i]);
    }

#else
    printf("%zu,%i,%i,%i,%i,%f,%f,%f,%f\n", DIM, dim_grid.x, dim_grid.y, dim_block.x, dim_block.y, bandwidths[0], bandwidths[1], bandwidths[2], bandwidths[3]);
#endif

    // CLEANING
    DELETE_CUDA_TIMER;
    free(data_cpu);
    free(reference);
    hipFree(data_gpu);
    hipFree(result_gpu);

    return EXIT_SUCCESS;
}
